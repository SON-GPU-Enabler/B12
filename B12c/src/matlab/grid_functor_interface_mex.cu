
#include <mex.h>
#include "matrix.h"

#include <thrust/functional.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/discard_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/transform.h>

#include "preprocessor_definitions.h"

#include "ThrustSystem.h"
#include "Grids.h"
#include "Functors.h"


void mexFunction(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[])
{
  // prhs[0] ... command string
  // prhs[1] ... grid type as string
  // prhs[2] ... dimension
  // prhs[3] ... input arguments for constructors
  // prhs[4] ... other input arguments
  
  // Get the command string
  char command[64];
  if (nrhs < 1 || mxGetString(prhs[0], command, sizeof(command) + 1)) {
    mexErrMsgTxt("First input should be a command string less than 64 characters long.");
  }
  if (nrhs < 2) {
    mexErrMsgTxt("Second input should be a string containing the grid type.");
  }
  if (nrhs < 3) {
    mexErrMsgTxt("Third input should be the dimension.");
  }
  if (nrhs < 4) {
    mexErrMsgTxt("Fourth input should be a vector containing the input arguments for the constructors.");
  }
  
  char gridType[255];
  mxGetString(prhs[1], gridType, sizeof(gridType) + 1);
  
  uint64_t DIM = *((uint64_t *) mxGetData(prhs[2]));
  
  char REAL[255];
  
  uint64_t * gridParams = (uint64_t *) mxGetData(prhs[3]);
  int nGridParams = mxGetNumberOfElements(prhs[3]);
  
  // points
  if (!strcmp("points", command)) {
    if (*((bool *) mxGetData(prhs[5]))) {
      // Allocate for output argument
      plhs[0] = mxCreateDoubleMatrix(DIM, mxGetM(prhs[4]), mxREAL);
      strcpy(REAL, "double");
    } else {
      // Allocate for output argument
      plhs[0] = mxCreateNumericMatrix(DIM, mxGetM(prhs[4]), mxSINGLE_CLASS, mxREAL);
      strcpy(REAL, "float");
    }
    char parallelMethod[255];
    mxGetString(prhs[6], parallelMethod, sizeof(parallelMethod) + 1);
    if (0) {
#if USE_CPP != -1
    } else if (!strcmp("cpu", parallelMethod)) {
      if (0) {
      }
#define LOCAL_DIMS_REALS_GRIDS_MACRO(dim, real, grid, n_grid_params)                                    \
  thrust::transform(typename b12::ThrustSystem<b12::CPP>::execution_policy(),                           \
                    (uint64_t *) mxGetData(prhs[4]), (uint64_t *) mxGetData(prhs[4]) + mxGetM(prhs[4]), \
                    thrust::make_transform_iterator(                                                    \
                        thrust::make_transform_iterator(                                                \
                            thrust::make_counting_iterator(b12::NrPoints(0)),                           \
                            thrust::placeholders::_1 * b12::NrPoints(DIM)),                             \
                        b12::AdvanceRealPointerFunctor<real>(                                           \
                            (real *) mxGetData(plhs[0]))),                                              \
                    thrust::make_discard_iterator(),                                                    \
                    b12::grid<dim, real>(N_PARAMS(n_grid_params, gridParams)));
ALL_ELSE_IF_COMBINATIONS_OF_DIMS_REALS_GRIDS
#undef LOCAL_DIMS_REALS_GRIDS_MACRO
      else {
        mexErrMsgTxt("Invalid grid scheme, e.g. wrong grid name or number of parameters, or dimension.");
      }
#endif
#if USE_OMP != -1
    } else if (!strcmp("omp", parallelMethod)) {
      if (0) {
      }
#define LOCAL_DIMS_REALS_GRIDS_MACRO(dim, real, grid, n_grid_params)                                    \
  thrust::transform(typename b12::ThrustSystem<b12::OMP>::execution_policy(),                           \
                    (uint64_t *) mxGetData(prhs[4]), (uint64_t *) mxGetData(prhs[4]) + mxGetM(prhs[4]), \
                    thrust::make_transform_iterator(                                                    \
                        thrust::make_transform_iterator(                                                \
                            thrust::make_counting_iterator(b12::NrPoints(0)),                           \
                            thrust::placeholders::_1 * b12::NrPoints(DIM)),                             \
                        b12::AdvanceRealPointerFunctor<real>(                                           \
                            (real *) mxGetData(plhs[0]))),                                              \
                    thrust::make_discard_iterator(),                                                    \
                    b12::grid<dim, real>(N_PARAMS(n_grid_params, gridParams)));
ALL_ELSE_IF_COMBINATIONS_OF_DIMS_REALS_GRIDS
#undef LOCAL_DIMS_REALS_GRIDS_MACRO
      else {
        mexErrMsgTxt("Invalid grid scheme, e.g. wrong grid name or number of parameters, or dimension.");
      }
#endif
#if USE_TBB != -1
    } else if (!strcmp("tbb", parallelMethod)) {
      if (0) {
      }
#define LOCAL_DIMS_REALS_GRIDS_MACRO(dim, real, grid, n_grid_params)                                    \
  thrust::transform(typename b12::ThrustSystem<b12::TBB>::execution_policy(),                           \
                    (uint64_t *) mxGetData(prhs[4]), (uint64_t *) mxGetData(prhs[4]) + mxGetM(prhs[4]), \
                    thrust::make_transform_iterator(                                                    \
                        thrust::make_transform_iterator(                                                \
                            thrust::make_counting_iterator(b12::NrPoints(0)),                           \
                            thrust::placeholders::_1 * b12::NrPoints(DIM)),                             \
                        b12::AdvanceRealPointerFunctor<real>(                                           \
                            (real *) mxGetData(plhs[0]))),                                              \
                    thrust::make_discard_iterator(),                                                    \
                    b12::grid<dim, real>(N_PARAMS(n_grid_params, gridParams)));
ALL_ELSE_IF_COMBINATIONS_OF_DIMS_REALS_GRIDS
#undef LOCAL_DIMS_REALS_GRIDS_MACRO
      else {
        mexErrMsgTxt("Invalid grid scheme, e.g. wrong grid name or number of parameters, or dimension.");
      }
#endif
#if USE_CUDA != -1
    } else if (!strcmp("gpu", parallelMethod)) {
      if (0) {
      }
#define LOCAL_DIMS_REALS_GRIDS_MACRO(dim, real, grid, n_grid_params)                 \
  typename b12::ThrustSystem<b12::CUDA>::Vector<uint64_t> inds(                      \
    (uint64_t *) mxGetData(prhs[4]),                                                 \
    (uint64_t *) mxGetData(prhs[4]) + mxGetM(prhs[4]));                              \
  typename b12::ThrustSystem<b12::CUDA>::Vector<real> result(DIM * mxGetM(prhs[4])); \
  thrust::transform(typename b12::ThrustSystem<b12::CUDA>::execution_policy(),       \
                    inds.begin(), inds.end(),                                        \
                    thrust::make_transform_iterator(                                 \
                        thrust::make_transform_iterator(                             \
                            thrust::make_counting_iterator(b12::NrPoints(0)),        \
                            thrust::placeholders::_1 * b12::NrPoints(DIM)),          \
                        b12::AdvanceRealPointerFunctor<real>(                        \
                            thrust::raw_pointer_cast(result.data()))),               \
                    thrust::make_discard_iterator(),                                 \
                    b12::grid<dim, real>(N_PARAMS(n_grid_params, gridParams)));      \
  thrust::copy(result.begin(), result.end(), (real *) mxGetData(plhs[0]));
ALL_ELSE_IF_COMBINATIONS_OF_DIMS_REALS_GRIDS
#undef LOCAL_DIMS_REALS_GRIDS_MACRO
      else {
        mexErrMsgTxt("Invalid grid scheme, e.g. wrong grid name or number of parameters, or dimension.");
      }
#endif
    } else {
      mexErrMsgTxt("Invalid parallel method.");
    }
    return;
  }
  
  // nPointsPerBox
  if (!strcmp("nPointsPerBox", command)) {
    // output for output argument
    plhs[0] = mxCreateNumericMatrix(1, 1, mxUINT64_CLASS, mxREAL);
    uint64_t * pr = (uint64_t *) mxGetData(plhs[0]);
    strcpy(REAL, "double");
    if (0) {
    }
#define LOCAL_DIMS_REALS_GRIDS_MACRO(dim, real, grid, n_grid_params) \
  pr[0] = b12::grid<dim, real>(N_PARAMS(n_grid_params, gridParams)).getNumberOfPointsPerBox();
ALL_ELSE_IF_COMBINATIONS_OF_DIMS_REALS_GRIDS
#undef LOCAL_DIMS_REALS_GRIDS_MACRO
    else {
      strcpy(REAL, "float");
      if (0) {
      }
#define LOCAL_DIMS_REALS_GRIDS_MACRO(dim, real, grid, n_grid_params) \
  pr[0] = b12::grid<dim, real>(N_PARAMS(n_grid_params, gridParams)).getNumberOfPointsPerBox();
ALL_ELSE_IF_COMBINATIONS_OF_DIMS_REALS_GRIDS
#undef LOCAL_DIMS_REALS_GRIDS_MACRO
      else {
        mexErrMsgTxt("Invalid grid scheme, e.g. wrong grid name or number of parameters, or dimension.");
      }
    }
    return;
  }
  
  // Got here, so command not recognized
  mexErrMsgTxt("Command not recognized.");
}
